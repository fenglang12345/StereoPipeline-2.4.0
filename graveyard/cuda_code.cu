#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

/* --------------------------- target code ------------------------------*/  
#define THREAD_COUNT 128

struct params { 
    float *input; 
    float *output; 
    int n; 
}; 
 
__global__ void flip_main (struct params p) 
{ 
  int i; 
  for (i = threadIdx.x; i < p.n; i += THREAD_COUNT) { 
    p.output[i] = 1.0f - p.input[i]; 
  } 
} 
 
/* --------------------------- host code ------------------------------*/ 

// Invert a 1 channel, 32-bit floating point image.
void invert_image (float* img, int width, int height) { 
    hipError_t         cudaStat; 
    float*              out = 0; 
    float*              in = 0; 
    struct params   funcParams; 
    int N = width * height;
 
    cout << "Allocating GPU Memory.\n";
    cudaStat = hipMalloc ((void **)&in, N * sizeof(in[0])); 
    cudaStat = hipMalloc ((void **)&out, N * sizeof(out[0])); 

    cout << "Copying data.\n";
    cudaStat = hipMemcpy (in, img, N * sizeof(img[0]), hipMemcpyHostToDevice); 
 
    funcParams.output = out; 
    funcParams.input = in; 
    funcParams.n = N; 
     
    cout << "Running kernel.\n";
    flip_main<<<1,THREAD_COUNT>>>(funcParams); 

    cout << "Copying result.\n";
    cudaStat = hipMemcpy (img, out, N * sizeof(out[0]), hipMemcpyDeviceToHost); 
} 


//------------

  /// Base class from which specific image resources derive.
  class CudaImageResource { 
    float* m_buffer;
    ImageFormat m_format;

  public:
  
    CudaImageResource(ImageFormat format): 
      m_format(format) {
      int32 size = m_format.cols * m_format.rows * m_format.planes;
      cudaStat = hipMalloc ((void **)&m_buffer, size * sizeof(float)); 
    }

    virtual ~CudaImageResource() {
      hipFree(m_buffer);
    };

    /// Returns the number of columns in an image resource.
    virtual int32 cols() const { return m_cols; }

    /// Returns the number of rows in an image resource.
    virtual int32 rows() const { return m_rows; }

    /// Returns the number of planes in an image resource.
    virtual int32 planes() const { return m_planes; }

    /// Returns the number of channels in a image resource.
    int32 channels() const { return num_channels( pixel_format() ); }

    /// Returns the native pixel format of the resource.
    virtual PixelFormatEnum pixel_format() const { return m_format.pixel_format; }

    /// Returns the native channel type of the resource.
    virtual ChannelTypeEnum channel_type() const { return m_format.channel_type; }

    /// Read the image resource at the given location into the given buffer.
    virtual void read( ImageBuffer const& buf, BBox2i const& bbox ) const {
      cudaStat = hipMemcpy (img, out, N * sizeof(out[0]), hipMemcpyDeviceToHost); 
    }

    /// Write the given buffer to the image resource at the given location.
    virtual void write( ImageBuffer const& buf, BBox2i const& bbox ) {
      cudaStat = hipMemcpy (in, img, N * sizeof(img[0]), hipMemcpyHostToDevice); 
    }

    /// Returns the optimal block size/alignment for partial reads or writes.
    virtual Vector2i native_block_size() const { return Vector2i(cols(),rows()); }

    /// Force any changes to be written to the resource.
    virtual void flush() {}

  };
